#include "hip/hip_runtime.h"
#include<iostream>
#include<stdio.h>
#include<string>
#include<cmath>
#include<hiprand.h>
#include<hiprand/hiprand_kernel.h>
#include"util.h"

#include "VIBE-NaiveParallel_Overlap-CUDA-kernel.h"

#define COLOR_BACKGROUND 0
#define COLOR_FOREGROUND 255


/* Function cuda_init_rand
 * Input: seed for the number generator
 *        state - an array of them already allocated on device
 *        cols - the width of our 2D data space
 */
__global__ void cuda_init_rand(unsigned long long seed,
                               hiprandState_t *state,
                               int cols){

  int j = blockIdx.x*blockDim.x + threadIdx.x;
  int i = blockIdx.y*blockDim.y + threadIdx.y;

  hiprand_init(seed, // the seed 
              (unsigned long long)i+j,  // the sequence 
              0,    //the offset 
              &state[i*cols + j]);
  
}

/* Function cuda_init_rand_wrapper
 * input: A seed to get the randoma number generator started.
 *        A pointer to an array of states that have been already
 *          allocated on the device
 *        The number of columns in our 2D data layout 
 *        The grid and block configuration
 */
extern "C" void cuda_init_rand_wrapper(unsigned long long seed,
                                       hiprandState_t *state,
                                       int cols, dim3 grid, dim3 block){
    (cuda_init_rand<<<grid,block>>>(seed,state,cols));
    if ( hipSuccess != hipGetLastError()){
          printf( "Error in hiprand init!\n" );
    }
}         


__global__ void segment_frames(unsigned char* map,pixel* frame,
                               pixel* init_model,int rows,int cols,
                               int samples,int radius,int match_threshold)
{
   bool match = false;
   int frame_index=0;
   int model_index=0;
   int dist = 0;
   int count = 0;
   int index = 0;
   int dist_threshold = 4.5*radius;

   pixel* pix_pointer;

   int j = blockIdx.x*blockDim.x + threadIdx.x;
   int i = blockIdx.y*blockDim.y + threadIdx.y;

        //index of each pixel
        if(i < rows && j < cols)
       {
         frame_index = i*cols + j;
         pix_pointer = &frame[frame_index];

         //index of model
         model_index = i*samples*cols + j*samples;
   
       while(count < match_threshold &&  index < samples)
        {

          

          dist =  abs((pix_pointer->b - init_model[model_index].b)) +
                  abs((pix_pointer->g -  init_model[model_index].g)) +
                  abs((pix_pointer->r -  init_model[model_index].r));

           if(dist < dist_threshold)
          {
             count++;
          }

          model_index += 1;
          index++;
       }

        if(count >= match_threshold)
       {
         match = true;
       }
        else
       {
         match =  false;
       }
       
        // creating the map by segmenting each pixel as foreground or 
        // background
       if(match)
         {
          map[i*cols + j] = COLOR_BACKGROUND;
          //cout <<  (int)map[120*cols + 160] << endl;        
         }
       else
         {
           map[i*cols + j] = COLOR_FOREGROUND;
         // cout << "not background" << endl;          
         }
     }

}

__global__ void update_model(unsigned char* segmentmap,
                             pixel* c_image,pixel* init_model,
                             hiprandState_t *state,
                             int rows,int cols,int time_sample,
                             int numsamples,int frames){

    int map_index;
    int neigh_index;
    int x,y,n;
    int time_stamp;

    
    int j = blockIdx.x*blockDim.x + threadIdx.x;
    int i = blockIdx.y*blockDim.y + threadIdx.y;
     
    
    map_index = i*cols + j;
    
    time_stamp = (u_int)hiprand(&state[i*cols + j])%time_sample;
    if(time_stamp == 0){

     if(segmentmap[map_index] == 0){
           
      int modelsample_index;
      int global_model_index;

      // which value in the model to update (0-19)
      modelsample_index = (u_int)hiprand(&(state[i*cols+j]))%numsamples;
         
      // memory index of the pixel + the model offset 
      global_model_index = map_index*numsamples + modelsample_index;

      // set the model value to the current pixel value  
      init_model[global_model_index] = c_image[map_index];

      n = (u_int)hiprand(&state[i*cols+j])%9; 
      x = n/3 - 1;
      y = n%3 - 1;

      if( i > 0 && i < rows -1 && j > 0 && j < cols - 1)
      {
        neigh_index = (x+i)*cols + (y+j);
        global_model_index = neigh_index*numsamples + modelsample_index;
        init_model[global_model_index] = c_image[map_index];
      }
    }
  }
}

extern "C" void cuda_segment_frames(unsigned char* map,pixel* frame,
                                   pixel* init_model,int rows,int cols,
                                   int samples,int radius,int match_threshold,
                                   dim3 grid,dim3 block)
{
      segment_frames<<<grid, block>>>(map,frame,
                                      init_model,rows,
                                      cols,samples,radius,
                                      match_threshold);
      if ( hipSuccess != hipGetLastError()){
          printf( "Error in segment kernel!\n" );
      }
  
}
extern "C" void cuda_update_model(unsigned char* map,pixel* frame,
                                  pixel* init_model, hiprandState_t *state,
                                  int rows,int cols,int time_samples,
                                  int modelsamples,int framecount,
                                  dim3 grid,dim3 block) {

     update_model<<<grid,block>>>(map,frame,init_model,state,
                                rows,cols,time_samples,
                                modelsamples,framecount);
     if ( hipSuccess != hipGetLastError()){
          printf( "Error in update kernel!\n" );
     }


}

                                      
