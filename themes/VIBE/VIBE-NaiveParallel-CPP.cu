#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <string>
#include <cmath>

#include "opencv2/opencv.hpp"

#include <stdlib.h>
#include <cstdlib>
#include <time.h>
#include <hip/hip_runtime.h>


#include "timer.cu"
#include "VIBE_Configuration.h"
#include "../common/Configuration.h"
#include "util.h"


using namespace std;
using namespace cv;

int iter = 0 ;

#define COLOR_BACKGROUND 0
#define COLOR_FOREGROUND 255

#define image(n,i,j,k) image[(n*(width*height) + i*(width) + j)*3 + k]
#define segmap(i,j)  segmap[(i*width) + j]


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
   
__global__ void segment_frames(unsigned char* map,pixels* frame,
                               pixels* init_model,int rows,int cols,
                               int samples,int radius,int match_threshold)
{
   bool match = false;
   int frame_index=0;
   int model_index=0;
   int dist = 0;
   int count = 0;
   int index = 0;

   pixels* pix_pointer;
   
   int j = blockIdx.x*blockDim.x + threadIdx.x;
   int i = blockIdx.y*blockDim.y + threadIdx.y; 
        
        //index of each pixel
        if(i < rows && j < cols)
       { 
         frame_index = i*cols + j;
         pix_pointer = &frame[frame_index];
       
         //index of model
         model_index = i*samples*cols + j*samples;
       
         // calculate the samples that lie within the radius threshold 
       
         while(count <= match_threshold &&  index < samples)
        {

          model_index =+ 1;
         
          dist =  abs((pix_pointer->b - init_model[model_index].b)*(pix_pointer->b-init_model[model_index].b)) + 
                  abs((pix_pointer->g-init_model[model_index].g)*(pix_pointer->g-init_model[model_index].g)) + 
                  abs((pix_pointer->r-init_model[model_index].r)*(pix_pointer->r-init_model[model_index].r));      
           
           if(dist < radius*radius)
          {
             count++;
          }    

          index++;
       }  

        if(count >= match_threshold)
       {
         match = true;
       }
        else
       {
         match =  false;
       }
         
        // creating the map by segmenting each pixel as foreground or 
        // background
       if(match)
         {
          map[i*cols + j] = COLOR_BACKGROUND;
          //cout <<  (int)map[120*cols + 160] << endl;        
         }
       else
         {
           map[i*cols + j] = COLOR_FOREGROUND;
         // cout << "not background" << endl;          
         }  
              
       }
  
}                      


int main(int argc,char* argv[])
{   


  std::string filename;
  int N;
  int R;
  int time_sample;
  int match_count;

//timer to calculate the initialization of the model
  GpuTimer timer1;

//timer to calculate the time taken to segment 
//the frames
  GpuTimer timer2;

//timer taken to calculate the time taken to update
// the model
  GpuTimer timer3;

//timer taken to calculate the total time
  GpuTimer timer4;
  
  float time1 = 0;
  float time2 = 0;
  float time3 = 0;
  float time4 = 0;

//Constructor for parsing command line arguments
  VIBE_Configuration config;

// Parse command line 
  config.parse(argc,argv);

  filename = config.getString("filename");
 
  if(filename == "NULL")
  {
    cout << " unable to find file" << endl;
    exit(-1);
  }
  

  N = config.getInt("numberofSamples");
  R = config.getInt("radiusofSphere");
  time_sample = config.getInt("timeSampling");
  match_count = config.getInt("matchingSamples");

//checking for command line arguments
  if(N < 0)
 {
   fprintf(stderr,"The value of N has to be a positive number %d\n",N);
   exit(-1);
 }

 if(R < 0)
 {
   fprintf(stderr,"The value of R has to be a positive number %d\n",R);
    exit(-1);
 } 

  if(time_sample < 0)
  {
   fprintf(stderr,"The value of time_sample has to be a positive number %d\n",time_sample);
   exit(-1);
  }

  if(match_count < 0)
  {
   fprintf(stderr,"The value of match_count has to be a positive number %d\n",match_count);
   exit(-1);
  }

//2 Initializiing pointers

  pixels *h_image = NULL;
  pixels *h_start_image = NULL;
  unsigned char *h_segmap = NULL;
  pixels *h_model = NULL;

  pixels *d_image = NULL;
  unsigned char *d_segmap = NULL;
  pixels *d_model = NULL;
  pixels *d_start_image = NULL;

// Parameters of the image
 
  int height;
  int width;
  int numofFrames;

// Initializing variables

  int frameCount = 0;
  int x_block,y_block; 

  x_block = config.getInt("bx");
  y_block = config.getInt("by"); 

// Getting frame info
  string& fp = filename;
  VideoCapture capture(fp);

//get frame info
  numofFrames = (int)capture.get(CV_CAP_PROP_FRAME_COUNT);
  width  = (int)capture.get(CV_CAP_PROP_FRAME_WIDTH);
  height = (int)capture.get(CV_CAP_PROP_FRAME_HEIGHT);
  cout << height << endl;
  cout << width << endl;
 
  
//Allocating memory to pixels
  size_t sizeModel = width*height*N;
  size_t sizeRGB = width*height;

  h_model = new pixels[sizeModel];
  h_image = new pixels[sizeRGB];
  h_start_image = h_image;


  const dim3 blockSize(x_block,y_block,1);
  const dim3 gridSize(1,1,1);
  //size_t dyn_mem_size = x_block*y_block*sizeof(pixels);

//Allocating memory to device memory
  gpuErrchk(hipMalloc((void**)&d_image,sizeRGB*sizeof(pixels)));
  gpuErrchk(hipMalloc((void**)&d_model,sizeModel*sizeof(pixels)));

//copying memory to device for model  
  gpuErrchk(hipMemcpy(d_model,h_model,sizeModel*sizeof(pixels),hipMemcpyHostToDevice));

// to create different random numbers every time
  srand(time(NULL));
   
// Reading all the frames and initializing the model 
  Mat frame;
  capture >> frame;

  timer4.Start();
  while(!frame.empty())
  {
 
// allocate memory for segmented map for each frame
    h_segmap = new unsigned char[width*height];

//allocate memory to device 
    gpuErrchk(hipMalloc((void**)&d_segmap,sizeRGB*sizeof(unsigned char))); 

//copy memory from host to device
    gpuErrchk(hipMemcpy(d_segmap,h_segmap,sizeRGB*sizeof(unsigned char),hipMemcpyHostToDevice));

// point image to start of space allocated    
    h_image = h_start_image;

// reading in data into image from capture instance frame
    for(int j=0;j < height*width;j++)
    {
       pixels pm ;
       pm.b = frame.data[3*j];
       pm.g = frame.data[3*j+1];
       pm.r = frame.data[3*j+2];
       h_image[j] = pm;
       
    }
// copy memory of image from host to device

    gpuErrchk(hipMemcpy(d_image,h_image,sizeRGB*sizeof(pixels),hipMemcpyHostToDevice));
    
// initialize pixel model and time 
    
    if(frameCount ==0){ 
     
      timer1.Start();
      initialize_model(h_image,h_model,height,width,N); 
      timer1.Stop();
      time1 = time1 = timer1.Elapsed(); 
    
    }

 
// segment the frame and time
   
   // timer2.Start();
    segment_frames<<<gridSize, blockSize>>>(d_segmap,d_image,
                                            d_model,height,
                                            width,N,R,
                                            match_count);

    timer2.Stop();
    time2 = time2 + timer2.Elapsed();
    
// update the model and its neighbour and time
  /*  if(frameCount > 0)
    {
      timer3.Start();      
      update_model(segmap,image,model,height,width,
                 time_sample,N,frameCount);
      timer3.Stop();
      time3  = time3 + timer3.Elapsed();    
    }*/

    gpuErrchk(hipMemcpy(h_model,d_model,sizeModel*sizeof(pixels),hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(h_segmap,d_segmap,sizeRGB*sizeof(unsigned char),hipMemcpyDeviceToHost));

    hipFree(d_segmap);
   
    frameCount++;
    cout <<"The frameCount is:" << frameCount <<  endl;
    
    delete[] h_segmap;
    
    capture >> frame;
   
  }
  timer4.Stop();
  time4 = time4 + timer4.Elapsed();
 
//total time for all frames


  hipFree(d_image);
  hipFree(d_model);

//delete the allocated arrays
  delete[] h_start_image;
  delete[] h_model;

//release the cpture instance
  capture.release();

//print the string configuration
  string lda = config.toLDAPString();
  cout << lda << endl;

  return 0;
}

